/***********************************************************************************
 *                                                                                 *
 * NAME: main.cu                                                                   *
 *                                                                                 *
 * AUTHOR: Michael Brockus.                                                        *
 *                                                                                 *
 * CONTACT: <mailto:michael@squidfarts.com>                                        *
 *                                                                                 *
 * NOTICES:                                                                        *
 *                                                                                 *
 * License: MIT                                                                    *
 *                                                                                 *
 ***********************************************************************************/

#include <hip/hip_runtime.h>
#include <iostream>



/*
 * This demonstrates a simple hello world program where the message
 * 'Hello, cuda' is desplayed to the standerd console.
 *
 */
int main(void)
{
    puts("Hello, cuda.");
    
    return EXIT_SUCCESS;
}// end of function main
